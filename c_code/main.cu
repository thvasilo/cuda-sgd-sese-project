#include "hip/hip_runtime.h"
#include "typedefs.cuh"
#include "sgd_io.cuh"
#include "sgd_thrust.cuh"
#include "sampling.cuh"
#include "testing.cuh"

#include <algorithm>
#include <cmath>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/random.h>
#include <thrust/inner_product.h>
#include <thrust/device_ptr.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <iostream>
#include <assert.h>

using namespace thrust::placeholders;

// Function to divide tasks up to threads
// Arguments: a: number of items to divide, b: desired number of threads in each block
int iDivUp(int a, int b){ return ((a % b) != 0) ? (a / b + 1) : (a / b); }


// Usage: Run with all arguments:
// args: [learning_rate] [iterations] [data_csv_file] [num_rows] [num_features] [batchsize]
// Setting batchsize to 0 uses the full data at each iteration
// NB: We are assuming that the csv has the format [features],[label]
// i.e. the last column is the label, and all others are features.
// num_features should equal the number of features only, i.e. the number of columns in the csv minus 1
// e.g.: > ./main 0.00001 10 data/5xy.csv 40 1 0
int main(int argc, char **argv) {

//	test_permutation();
//
//	return 0;

//	test_gemv();
//
//	test_matrix_scale();
//
//	test_col_sums();
//
//	return 0;

	if	(argc != 7) {
		std::cout << "usage: ./sgd_thrust.o [learning_rate] "
				"[iterations] [data_csv_file] [num_rows] [num_features] [batchsize]" << std::endl;
		return 1;
	}

	float learning_rate = atof(argv[1]);
	const int MAX_EPOCHS = atoi(argv[2]);
	const std::string filename = argv[3];
	const int R = atoi(argv[4]);
	const int C = atoi(argv[5]);
	const int batchsize = (atoi(argv[6])  == 0) ? R : atoi(argv[6]);
	const int num_batches = (int)std::floor(R/(float)batchsize);

	hipEvent_t start_memory;
	hipEvent_t stop_memory;

	// Create the events
	hipEventCreate(&start_memory);
	hipEventCreate(&stop_memory);

	// Start recording
	hipEventRecord(start_memory);
	// The number of threads we allocate per block
	const int THREADS_PER_BLOCK = batchsize;

	// Initialize data vector on host
	thrust_host_float data_h(R * C);

	// Initialize labels vector on host
	thrust_host_float labels_h(R);

	// Read data from csv file into host vectors
	read_csv(filename, data_h, labels_h, R, C);

	// Copy data from host vectors to device
	// note: d_vec.data() returns a device_ptr
	thrust_dev_float data_d = data_h;
	float * data_raw_ptr = thrust::raw_pointer_cast(data_d.data());
	thrust_dev_float labels_d = labels_h;
	float * labels_raw_ptr = thrust::raw_pointer_cast(labels_d.data());

	// Initialize weights
	thrust_dev_float weights(C);
	thrust::default_random_engine rng;
	thrust::uniform_real_distribution<float> weight_dist(0.0, 0.01);
	float * weights_raw_ptr = thrust::raw_pointer_cast(weights.data());
	for (size_t i = 0; i < weights.size(); i++) {
				weights[i] = weight_dist(rng);
	}

	// Initialize gradients
	thrust_dev_float gradients(batchsize * C);
	float * gradients_raw_ptr = thrust::raw_pointer_cast(gradients.data());

	// Initialize loss derivative vector
	thrust_dev_float loss_derivative(batchsize);
	float * loss_derivative_raw_ptr = thrust::raw_pointer_cast(loss_derivative.data());

	//Initialize errors vector
	thrust_dev_float errors(R);
	float * errors_raw_ptr = thrust::raw_pointer_cast(errors.data());

	// Allocate storage for row sums and indices
	thrust_dev_float col_sums(C);

	// Allocate storage for matrix and vector shuffled copies.
	thrust_dev_float data_shuffled_d(R*C);
	float * data_shuffled_raw_ptr = thrust::raw_pointer_cast(data_shuffled_d.data());
	thrust_dev_float labels_shuffled_d(R);
	float * labels_shuffled_raw_ptr = thrust::raw_pointer_cast(labels_shuffled_d.data());

	// Initialize batch indices vector
	thrust::device_vector<unsigned> batch_indices_d(R);
	// Fill indices vector, we first create and index vector, shuffle it and copy to device vector
	std::vector<unsigned> ind_vector(R);
	for (int i = 0; i < R; ++i) {
		ind_vector[i] = i;
	}
	// Shuffle the vector on the host, and copy to the device
	std::random_shuffle(ind_vector.begin(), ind_vector.end());
	batch_indices_d = ind_vector;

	// Now measure the differences
	hipEventRecord(stop_memory);
	hipEventSynchronize(stop_memory);
	float miliseconds_memory = 0;
	hipEventElapsedTime(&miliseconds_memory, start_memory, stop_memory);
	printf("Memory time = %f ms\n", miliseconds_memory);

	hipEventDestroy(start_memory);
	hipEventDestroy(stop_memory);

	hipEvent_t start;
	hipEvent_t stop;

	// Create the events
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	// Start measuring the event
	hipEventRecord(start);

	for (int epoch = 1; epoch <= MAX_EPOCHS; ++epoch) {
		// We shuffle the data indexes before the start of each epoch on the host, and copy to the GPU
		std::random_shuffle ( ind_vector.begin(), ind_vector.end());
		// Currently we are shuffling the indices vector on host and copying to device.
		// Maybe it is possible to create a random permutation vector on the GPU, or shuffle an existing one (i.e. copy from device
		// only once.)
		batch_indices_d = ind_vector;

		// This creates a permutation of the data and copies it to data_shuffled_d and the same with labels.
		// TODO: Is there a way to avoid having the dev-dev copy at each iteration?
		permute_data_and_labels(
						data_d,
						labels_d,
						batch_indices_d,
						data_shuffled_d,
						labels_shuffled_d,
						R,
						C);
		for (int batch = 0; batch < num_batches; ++batch) {

			// Reset gradients and errors
			thrust::fill(gradients.begin(), gradients.end(), 0.0); // TODO: Necessary?
			//thrust::fill(loss_derivative.begin(), loss_derivative.end(), 0.0);


			// Pointer offsets to be consistent with current batch
			int offset = batch * batchsize;
			float * cur_batch_data_ptr = data_shuffled_raw_ptr + offset;
			thrust::device_ptr<float> cur_batch_data_dev_ptr(cur_batch_data_ptr);
			float * cur_batch_labels_ptr = labels_shuffled_raw_ptr + offset;

			// Calculate the loss derivative vector
			calculate_loss_derivative_cublas(
					cur_batch_data_ptr,
					cur_batch_labels_ptr,
					weights_raw_ptr,
					loss_derivative_raw_ptr,
					R,
					C,
					batchsize);

//			print_vector(loss_derivative, "loss_derivative");

			// The gradient matrix is equal to the feature matrix of the batch scaled by the loss derivative vector
			// TODO: Can we fuse some of the following operations? The column sum and and scaling could be fused no?
			scale_matrix_rows_by_vector(
				cur_batch_data_dev_ptr,
				loss_derivative,
				gradients, // Result stored in gradient matrix of size batchsize*C
				batchsize,
				C);

//			print_matrix(gradients, "gradients", batchsize, C);

			// Once we have the scaled data matrix, i.e. the gradients we need to sum the columns and scale to get
			// the avg. gradient vector.
			calculate_column_sums(
				gradients_raw_ptr,
				col_sums, // col_sums will now contain the sum of the columns in the gradient matrix
				batchsize,
				C);

//			print_vector(col_sums, "gradients_col_sums");
			// Scale gradient sum vector to obtain avg. gradient vector
			thrust::for_each(col_sums.begin(), col_sums.end(), _1 / (float)batchsize);

			//Update the weight vector
			float a = -(learning_rate / std::pow(epoch, 0.25));

			// Thrust SAXPY, used to update the weights vector
			thrust::transform(col_sums.begin(), col_sums.end(),  // input range #1
					weights.begin(),           // input range #2
					weights.begin(),           // output range
					a * _1 + _2);        // placeholder expression
		}
		if	(epoch % 100 == 0) {
			thrust::fill(errors.begin(), errors.end(), 0.0);
			// Calculate the squared error for each data point
			squared_errors<<<iDivUp(R, THREADS_PER_BLOCK), THREADS_PER_BLOCK>>>(
					data_raw_ptr,
					labels_raw_ptr,
					weights_raw_ptr,
					errors_raw_ptr,
					R,
					C);
			// Reduce/sum the errors
			float sq_err_sum = thrust::reduce(errors.begin(), errors.end());
		}

	}


	// Print final weights and squared error sum
	// Calculate the squared error for each data point
	squared_errors<<<iDivUp(R, THREADS_PER_BLOCK), THREADS_PER_BLOCK>>>(
			data_raw_ptr,
			labels_raw_ptr,
			weights_raw_ptr,
			errors_raw_ptr,
			R,
			C);

	// Print final quantities
	float sq_err_sum = thrust::reduce(errors.begin(), errors.end());
	std::cout << "Squared error sum: " << sq_err_sum << std::endl;
	print_vector(weights, "weights");	
	
	// Get the second time
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float miliseconds = 0;
	hipEventElapsedTime(&miliseconds, start, stop);
	printf("kernel time = %f ms\n", miliseconds);


	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}
